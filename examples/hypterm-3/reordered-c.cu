#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void hypterm_0 (double * __restrict__ flux_in_0, double * __restrict__ flux_in_1, double * __restrict__ flux_in_2, double * __restrict__ flux_in_3, double * __restrict__ cons_in_1, double * __restrict__ cons_in_2, double * __restrict__ cons_in_3, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.z);

	double (*flux_0)[308][308] = (double (*)[308][308])flux_in_0;
	double (*flux_1)[308][308] = (double (*)[308][308])flux_in_1;
	double (*flux_2)[308][308] = (double (*)[308][308])flux_in_2;
	double (*flux_3)[308][308] = (double (*)[308][308])flux_in_3;
	double (*cons_1)[308][308] = (double (*)[308][308])cons_in_1;
	double (*cons_2)[308][308] = (double (*)[308][308])cons_in_2;
	double (*cons_3)[308][308] = (double (*)[308][308])cons_in_3;
    double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
    double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
    double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
    double (*q_4)[308][308] = (double (*)[308][308])q_in_4;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
double _t_1_ = cons_1[k][j][i+1];
_t_1_ -= cons_1[k][j][i-1];
double _t_0_ = 0.8 * _t_1_;
double _t_2_ = cons_1[k][j][i+2];
_t_2_ -= cons_1[k][j][i-2];
_t_0_ -= 0.2 * _t_2_;
double _t_3_ = cons_1[k][j][i+3];
_t_3_ -= cons_1[k][j][i-3];
_t_0_ += 0.038 * _t_3_;
double _t_4_ = cons_1[k][j][i+4];
_t_4_ -= cons_1[k][j][i-4];
_t_0_ -= 0.0035 * _t_4_;
double flux_0kc0jc0ic0 = _t_0_ * dxinv0;

double _t_12_ = cons_1[k][j][i+1] * q_1[k][j][i+1];
_t_12_ -= cons_1[k][j][i-1] * q_1[k][j][i-1];
_t_12_ += q_4[k][j][i+1];
_t_12_ -= q_4[k][j][i-1];
double _t_11_ = 0.8 * _t_12_;
double _t_13_ = cons_1[k][j][i+2] * q_1[k][j][i+2];
_t_13_ -= cons_1[k][j][i-2] * q_1[k][j][i-2];
_t_13_ += q_4[k][j][i+2];
_t_13_ -= q_4[k][j][i-2];
_t_11_ -= 0.2 * _t_13_;
double _t_14_ = cons_1[k][j][i+3] * q_1[k][j][i+3];
_t_14_ -= cons_1[k][j][i-3] * q_1[k][j][i-3];
_t_14_ += q_4[k][j][i+3];
_t_14_ -= q_4[k][j][i-3];
_t_11_ += 0.038 * _t_14_;
double _t_15_ = cons_1[k][j][i+4] * q_1[k][j][i+4];
_t_15_ -= cons_1[k][j][i-4] * q_1[k][j][i-4];
_t_15_ += q_4[k][j][i+4];
_t_15_ -= q_4[k][j][i-4];
_t_11_ -= 0.0035 * _t_15_;
double flux_1kc0jc0ic0 = _t_11_ * dxinv0;

double _t_23_ = cons_2[k][j][i+1] * q_1[k][j][i+1];
_t_23_ -= cons_2[k][j][i-1] * q_1[k][j][i-1];
double _t_22_ = 0.8 * _t_23_;
double _t_24_ = cons_2[k][j][i+2] * q_1[k][j][i+2];
_t_24_ -= cons_2[k][j][i-2] * q_1[k][j][i-2];
_t_22_ -= 0.2 * _t_24_;
double _t_25_ = cons_2[k][j][i+3] * q_1[k][j][i+3];
_t_25_ -= cons_2[k][j][i-3] * q_1[k][j][i-3];
_t_22_ += 0.038 * _t_25_;
double _t_26_ = cons_2[k][j][i+4] * q_1[k][j][i+4];
_t_26_ -= cons_2[k][j][i-4] * q_1[k][j][i-4];
_t_22_ -= 0.0035 * _t_26_;
double flux_2kc0jc0ic0 = _t_22_ * dxinv0;

double _t_34_ = cons_3[k][j][i+1] * q_1[k][j][i+1];
_t_34_ -= cons_3[k][j][i-1] * q_1[k][j][i-1];
double _t_33_ = 0.8 * _t_34_;
double _t_35_ = cons_3[k][j][i+2] * q_1[k][j][i+2];
_t_35_ -= cons_3[k][j][i-2] * q_1[k][j][i-2];
_t_33_ -= 0.2 * _t_35_;
double _t_36_ = cons_3[k][j][i+3] * q_1[k][j][i+3];
_t_36_ -= cons_3[k][j][i-3] * q_1[k][j][i-3];
_t_33_ += 0.038 * _t_36_;
double _t_37_ = cons_3[k][j][i+4] * q_1[k][j][i+4];
_t_37_ -= cons_3[k][j][i-4] * q_1[k][j][i-4];
_t_33_ -= 0.0035 * _t_37_;
double flux_3kc0jc0ic0 = _t_33_ * dxinv0;

double _t_7_ = cons_2[k][j+1][i];
_t_7_ -= cons_2[k][j-1][i];
double _t_6_ = 0.8 * _t_7_;
double _t_8_ = cons_2[k][j+2][i];
_t_8_ -= cons_2[k][j-2][i];
_t_6_ -= 0.2 * _t_8_;
double _t_9_ = cons_2[k][j+3][i];
_t_9_ -= cons_2[k][j-3][i];
_t_6_ += 0.038 * _t_9_;
double _t_10_ = cons_2[k][j+4][i];
_t_10_ -= cons_2[k][j-4][i];
_t_6_ -= 0.0035 * _t_10_;
flux_0kc0jc0ic0 -= _t_6_ * dxinv1;

double _t_29_ = cons_2[k][j+1][i] * q_2[k][j+1][i];
_t_29_ -= cons_2[k][j-1][i] * q_2[k][j-1][i];
_t_29_ += q_4[k][j+1][i];
_t_29_ -= q_4[k][j-1][i];
double _t_28_ = 0.8 * _t_29_;
double _t_30_ = cons_2[k][j+2][i] * q_2[k][j+2][i];
_t_30_ -= cons_2[k][j-2][i] * q_2[k][j-2][i];
_t_30_ += q_4[k][j+2][i];
_t_30_ -= q_4[k][j-2][i];
_t_28_ -= 0.2 * _t_30_;
double _t_31_ = cons_2[k][j+3][i] * q_2[k][j+3][i];
_t_31_ -= cons_2[k][j-3][i] * q_2[k][j-3][i];
_t_31_ += q_4[k][j+3][i];
_t_31_ -= q_4[k][j-3][i];
_t_28_ += 0.038 * _t_31_;
double _t_32_ = cons_2[k][j+4][i] * q_2[k][j+4][i];
_t_32_ -= cons_2[k][j-4][i] * q_2[k][j-4][i];
_t_32_ += q_4[k][j+4][i];
_t_32_ -= q_4[k][j-4][i];
_t_28_ -= 0.0035 * _t_32_;
flux_2kc0jc0ic0 -= _t_28_ * dxinv1;

double _t_18_ = cons_1[k][j+1][i] * q_2[k][j+1][i];
_t_18_ -= cons_1[k][j-1][i] * q_2[k][j-1][i];
double _t_17_ = 0.8 * _t_18_;
double _t_19_ = cons_1[k][j+2][i] * q_2[k][j+2][i];
_t_19_ -= cons_1[k][j-2][i] * q_2[k][j-2][i];
_t_17_ -= 0.2 * _t_19_;
double _t_20_ = cons_1[k][j+3][i] * q_2[k][j+3][i];
_t_20_ -= cons_1[k][j-3][i] * q_2[k][j-3][i];
_t_17_ += 0.038 * _t_20_;
double _t_21_ = cons_1[k][j+4][i] * q_2[k][j+4][i];
_t_21_ -= cons_1[k][j-4][i] * q_2[k][j-4][i];
_t_17_ -= 0.0035 * _t_21_;
flux_1kc0jc0ic0 -= _t_17_ * dxinv1;

double _t_40_ = cons_3[k][j+1][i] * q_2[k][j+1][i];
_t_40_ -= cons_3[k][j-1][i] * q_2[k][j-1][i];
double _t_39_ = 0.8 * _t_40_;
double _t_41_ = cons_3[k][j+2][i] * q_2[k][j+2][i];
_t_41_ -= cons_3[k][j-2][i] * q_2[k][j-2][i];
_t_39_ -= 0.2 * _t_41_;
double _t_42_ = cons_3[k][j+3][i] * q_2[k][j+3][i];
_t_42_ -= cons_3[k][j-3][i] * q_2[k][j-3][i];
_t_39_ += 0.038 * _t_42_;
double _t_43_ = cons_3[k][j+4][i] * q_2[k][j+4][i];
_t_43_ -= cons_3[k][j-4][i] * q_2[k][j-4][i];
_t_39_ -= 0.0035 * _t_43_;
flux_3kc0jc0ic0 -= _t_39_ * dxinv1;

flux_0[k][j][i] = flux_0kc0jc0ic0;
flux_1[k][j][i] = flux_1kc0jc0ic0;
flux_2[k][j][i] = flux_2kc0jc0ic0;
flux_3[k][j][i] = flux_3kc0jc0ic0;
	} 
}

__global__ void hypterm_1 (double * __restrict__ flux_in_0, double * __restrict__ flux_in_1, double * __restrict__ flux_in_2, double * __restrict__ flux_in_3, double * __restrict__ cons_in_1, double * __restrict__ cons_in_2, double * __restrict__ cons_in_3, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
    //Determing the block's indices
    int blockdim_i= (int)(blockDim.x);
    int i0 = (int)(blockIdx.x)*(blockdim_i);
    int i = max (i0, 0) + (int)(threadIdx.x);
    int blockdim_j= (int)(blockDim.y);
    int j0 = (int)(blockIdx.y)*(blockdim_j);
    int j = max (j0, 0) + (int)(threadIdx.y);
    int blockdim_k= (int)(blockDim.z);
    int k0 = (int)(blockIdx.z)*(4*blockdim_k);
    int k = max (k0, 0) + (int)(4*threadIdx.z);

    double (*flux_0)[308][308] = (double (*)[308][308])flux_in_0;
    double (*flux_1)[308][308] = (double (*)[308][308])flux_in_1;
    double (*flux_2)[308][308] = (double (*)[308][308])flux_in_2;
    double (*flux_3)[308][308] = (double (*)[308][308])flux_in_3;
    double (*cons_1)[308][308] = (double (*)[308][308])cons_in_1;
    double (*cons_2)[308][308] = (double (*)[308][308])cons_in_2;
    double (*cons_3)[308][308] = (double (*)[308][308])cons_in_3;
    double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
    double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
    double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
    double (*q_4)[308][308] = (double (*)[308][308])q_in_4;
	double flux0_a, flux1_a, flux2_a, flux3_a;
	double flux0_b, flux1_b, flux2_b, flux3_b;
	double flux0_c, flux1_c, flux2_c, flux3_c;
	double flux0_d, flux1_d, flux2_d, flux3_d;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
flux0_a = flux_0[k][j][i];
double flux_0kc0jc0ic0 = flux0_a;

double _t_1_ = cons_3[k+1][j][i];
_t_1_ -= cons_3[k-1][j][i];
double _t_0_ = 0.8 * _t_1_;
double _t_2_ = cons_3[k+2][j][i];
_t_2_ -= cons_3[k-2][j][i];
_t_0_ -= 0.2 * _t_2_;
double _t_3_ = cons_3[k+3][j][i];
_t_3_ -= cons_3[k-3][j][i];
_t_0_ += 0.038 * _t_3_;
double _t_4_ = cons_3[k+4][j][i];
_t_4_ -= cons_3[k-4][j][i];
_t_0_ -= 0.0035 * _t_4_;
flux_0kc0jc0ic0 -= _t_0_ * dxinv2;

flux0_b = flux_0[k+1][j][i];
double flux_0kp1jc0ic0 = flux0_b;

double _t_7_ = cons_3[k+3][j][i];
_t_7_ -= cons_3[k-1][j][i];
double _t_5_ = -(0.2 * _t_7_);
double _t_8_ = cons_3[k+4][j][i];
_t_8_ -= cons_3[k-2][j][i];
_t_5_ += 0.038 * _t_8_;
double _t_6_ = cons_3[k+2][j][i];
_t_6_ -= cons_3[k][j][i];
_t_5_ += 0.8 * _t_6_;
double _t_9_ = -(cons_3[k-3][j][i]);
_t_9_ += cons_3[k+5][j][i];
_t_5_ -= 0.0035 * _t_9_;
flux_0kp1jc0ic0 -= _t_5_ * dxinv2;

flux0_c = flux_0[k+2][j][i];
double flux_0kp2jc0ic0 = flux0_c;

double _t_11_ = cons_3[k+3][j][i];
_t_11_ -= cons_3[k+1][j][i];
double _t_10_ = 0.8 * _t_11_;
double _t_12_ = cons_3[k+4][j][i];
_t_12_ -= cons_3[k][j][i];
_t_10_ -= 0.2 * _t_12_;
double _t_13_ = cons_3[k+5][j][i];
_t_13_ -= cons_3[k-1][j][i];
_t_10_ += 0.038 * _t_13_;
double _t_14_ = -(cons_3[k-2][j][i]);
_t_14_ += cons_3[k+6][j][i];
_t_10_ -= 0.0035 * _t_14_;
flux_0kp2jc0ic0 -= _t_10_ * dxinv2;

flux0_d = flux_0[k+3][j][i];
double flux_0kp3jc0ic0 = flux0_d;

double _t_16_ = cons_3[k+4][j][i];
_t_16_ -= cons_3[k+2][j][i];
double _t_15_ = 0.8 * _t_16_;
double _t_17_ = cons_3[k+5][j][i];
_t_17_ -= cons_3[k+1][j][i];
_t_15_ -= 0.2 * _t_17_;
double _t_18_ = cons_3[k+6][j][i];
_t_18_ -= cons_3[k][j][i];
_t_15_ += 0.038 * _t_18_;
double _t_19_ = -(cons_3[k-1][j][i]);
_t_19_ += cons_3[k+7][j][i];
_t_15_ -= 0.0035 * _t_19_;
flux_0kp3jc0ic0 -= _t_15_ * dxinv2;

flux1_a = flux_1[k][j][i];
double flux_1kc0jc0ic0 = flux1_a;

double _t_24_ = -(cons_1[k-4][j][i] * q_3[k-4][j][i]);
_t_24_ += cons_1[k+4][j][i] * q_3[k+4][j][i];
double _t_20_ = -(0.0035 * _t_24_);
double _t_21_ = cons_1[k+1][j][i] * q_3[k+1][j][i];
_t_21_ -= cons_1[k-1][j][i] * q_3[k-1][j][i];
_t_20_ += 0.8 * _t_21_;
double _t_22_ = cons_1[k+2][j][i] * q_3[k+2][j][i];
_t_22_ -= cons_1[k-2][j][i] * q_3[k-2][j][i];
_t_20_ -= 0.2 * _t_22_;
double _t_23_ = cons_1[k+3][j][i] * q_3[k+3][j][i];
_t_23_ -= cons_1[k-3][j][i] * q_3[k-3][j][i];
_t_20_ += 0.038 * _t_23_;
flux_1kc0jc0ic0 -= _t_20_ * dxinv2;

flux1_b = flux_1[k+1][j][i];
double flux_1kp1jc0ic0 = flux1_b;
double _v_15_ = cons_1[k+3][j][i] * q_3[k+3][j][i];
double _v_16_ = cons_1[k-1][j][i] * q_3[k-1][j][i];
double _v_17_ = cons_1[k+4][j][i] * q_3[k+4][j][i];
double _v_18_ = cons_1[k-2][j][i] * q_3[k-2][j][i];
double _v_20_ = cons_1[k-3][j][i] * q_3[k-3][j][i];
double _v_13_ = cons_1[k+2][j][i] * q_3[k+2][j][i];
double _v_23_ = cons_1[k+1][j][i] * q_3[k+1][j][i];
double _v_47_ = cons_2[k-4][j][i] * q_3[k-4][j][i];
double _v_83_ = cons_3[k-4][j][i] * q_3[k-4][j][i];
double _v_82_ = cons_3[k+4][j][i] * q_3[k+4][j][i];
double _v_76_ = cons_3[k+1][j][i] * q_3[k+1][j][i];
double _v_77_ = cons_3[k-1][j][i] * q_3[k-1][j][i];
double _v_78_ = cons_3[k+2][j][i] * q_3[k+2][j][i];
double _v_79_ = cons_3[k-2][j][i] * q_3[k-2][j][i];
double _v_80_ = cons_3[k+3][j][i] * q_3[k+3][j][i];
double _v_81_ = cons_3[k-3][j][i] * q_3[k-3][j][i];

double _t_27_ = _v_15_;
_t_27_ -= _v_16_;
double _t_25_ = -(0.2 * _t_27_);
double _t_28_ = _v_17_;
_t_28_ -= _v_18_;
_t_25_ += 0.038 * _t_28_;
double _t_29_ = -(_v_20_);
_t_29_ += cons_1[k+5][j][i] * q_3[k+5][j][i];
_t_25_ -= 0.0035 * _t_29_;
double _t_26_ = _v_13_;
_t_26_ -= cons_1[k][j][i] * q_3[k][j][i];
_t_25_ += 0.8 * _t_26_;
flux_1kp1jc0ic0 -= _t_25_ * dxinv2;

flux1_c = flux_1[k+2][j][i];
double flux_1kp2jc0ic0 = flux1_c;
double _v_25_ = cons_1[k][j][i] * q_3[k][j][i];
double _v_26_ = cons_1[k+5][j][i] * q_3[k+5][j][i];
double _v_91_ = cons_3[k+5][j][i] * q_3[k+5][j][i];
double _v_86_ = cons_3[k][j][i] * q_3[k][j][i];

double _t_31_ = _v_15_;
_t_31_ -= _v_23_;
double _t_30_ = 0.8 * _t_31_;
double _t_32_ = _v_17_;
_t_32_ -= _v_25_;
_t_30_ -= 0.2 * _t_32_;
double _t_33_ = _v_26_;
_t_33_ -= _v_16_;
_t_30_ += 0.038 * _t_33_;
double _t_34_ = -(_v_18_);
_t_34_ += cons_1[k+6][j][i] * q_3[k+6][j][i];
_t_30_ -= 0.0035 * _t_34_;
flux_1kp2jc0ic0 -= _t_30_ * dxinv2;

flux1_d = flux_1[k+3][j][i];
double flux_1kp3jc0ic0 = flux1_d;
double _v_35_ = cons_1[k+6][j][i] * q_3[k+6][j][i];
double _v_100_ = cons_3[k+6][j][i] * q_3[k+6][j][i];

double _t_36_ = _v_17_;
_t_36_ -= _v_13_;
double _t_35_ = 0.8 * _t_36_;
double _t_37_ = _v_26_;
_t_37_ -= _v_23_;
_t_35_ -= 0.2 * _t_37_;
double _t_38_ = _v_35_;
_t_38_ -= _v_25_;
_t_35_ += 0.038 * _t_38_;
double _t_39_ = -(_v_16_);
_t_39_ += cons_1[k+7][j][i] * q_3[k+7][j][i];
_t_35_ -= 0.0035 * _t_39_;
flux_1kp3jc0ic0 -= _t_35_ * dxinv2;

flux2_a = flux_2[k][j][i];
double flux_2kc0jc0ic0 = flux2_a;
double _v_73_ = cons_2[k+7][j][i] * q_3[k+7][j][i];
double _v_109_ = cons_3[k+7][j][i] * q_3[k+7][j][i];

double _t_44_ = -(_v_47_);
double _v_46_ = cons_2[k+4][j][i] * q_3[k+4][j][i];
_t_44_ += _v_46_;
double _t_40_ = -(0.0035 * _t_44_);
double _v_40_ = cons_2[k+1][j][i] * q_3[k+1][j][i];
double _t_41_ = _v_40_;
double _v_41_ = cons_2[k-1][j][i] * q_3[k-1][j][i];
_t_41_ -= _v_41_;
_t_40_ += 0.8 * _t_41_;
double _v_42_ = cons_2[k+2][j][i] * q_3[k+2][j][i];
double _t_42_ = _v_42_;
double _v_43_ = cons_2[k-2][j][i] * q_3[k-2][j][i];
_t_42_ -= _v_43_;
_t_40_ -= 0.2 * _t_42_;
double _v_44_ = cons_2[k+3][j][i] * q_3[k+3][j][i];
double _t_43_ = _v_44_;
double _v_45_ = cons_2[k-3][j][i] * q_3[k-3][j][i];
_t_43_ -= _v_45_;
_t_40_ += 0.038 * _t_43_;
flux_2kc0jc0ic0 -= _t_40_ * dxinv2;

flux2_b = flux_2[k+1][j][i];
double flux_2kp1jc0ic0 = flux2_b;

double _t_47_ = _v_44_;
_t_47_ -= _v_41_;
double _t_45_ = -(0.2 * _t_47_);
double _t_48_ = _v_46_;
_t_48_ -= _v_43_;
_t_45_ += 0.038 * _t_48_;
double _t_49_ = -(_v_45_);
double _v_55_ = cons_2[k+5][j][i] * q_3[k+5][j][i];
_t_49_ += _v_55_;
_t_45_ -= 0.0035 * _t_49_;
double _t_46_ = _v_42_;
double _v_50_ = cons_2[k][j][i] * q_3[k][j][i];
_t_46_ -= _v_50_;
_t_45_ += 0.8 * _t_46_;
flux_2kp1jc0ic0 -= _t_45_ * dxinv2;

flux2_c = flux_2[k+2][j][i];
double flux_2kp2jc0ic0 = flux2_c;

double _t_51_ = _v_44_;
_t_51_ -= _v_40_;
double _t_50_ = 0.8 * _t_51_;
double _t_52_ = _v_46_;
_t_52_ -= _v_50_;
_t_50_ -= 0.2 * _t_52_;
double _t_53_ = _v_55_;
_t_53_ -= _v_41_;
_t_50_ += 0.038 * _t_53_;
double _t_54_ = -(_v_43_);
double _v_64_ = cons_2[k+6][j][i] * q_3[k+6][j][i];
_t_54_ += _v_64_;
_t_50_ -= 0.0035 * _t_54_;
flux_2kp2jc0ic0 -= _t_50_ * dxinv2;

flux2_d = flux_2[k+3][j][i];
double flux_2kp3jc0ic0 = flux2_d;

double _t_56_ = _v_46_;
_t_56_ -= _v_42_;
double _t_55_ = 0.8 * _t_56_;
double _t_57_ = _v_55_;
_t_57_ -= _v_40_;
_t_55_ -= 0.2 * _t_57_;
double _t_58_ = _v_64_;
_t_58_ -= _v_50_;
_t_55_ += 0.038 * _t_58_;
double _t_59_ = -(_v_41_);
_t_59_ += _v_73_;
_t_55_ -= 0.0035 * _t_59_;
flux_2kp3jc0ic0 -= _t_55_ * dxinv2;

flux3_a = flux_3[k][j][i];
double flux_3kc0jc0ic0 = flux3_a;

double _t_64_ = -(_v_83_);
_t_64_ += _v_82_;
_t_64_ -= q_4[k-4][j][i];
_t_64_ += q_4[k+4][j][i];
double _t_60_ = -(0.0035 * _t_64_);
double _t_61_ = _v_76_;
_t_61_ -= _v_77_;
_t_61_ += q_4[k+1][j][i];
_t_61_ -= q_4[k-1][j][i];
_t_60_ += 0.8 * _t_61_;
double _t_62_ = _v_78_;
_t_62_ -= _v_79_;
_t_62_ += q_4[k+2][j][i];
_t_62_ -= q_4[k-2][j][i];
_t_60_ -= 0.2 * _t_62_;
double _t_63_ = _v_80_;
_t_63_ -= _v_81_;
_t_63_ += q_4[k+3][j][i];
_t_63_ -= q_4[k-3][j][i];
_t_60_ += 0.038 * _t_63_;
flux_3kc0jc0ic0 -= _t_60_ * dxinv2;

flux3_b = flux_3[k+1][j][i];
double flux_3kp1jc0ic0 = flux3_b;
double _t_69_ = -(q_4[k-3][j][i]);

_t_69_ -= _v_81_;
_t_69_ += _v_91_;
_t_69_ += q_4[k+5][j][i];
double _t_65_ = -(0.0035 * _t_69_);
double _t_67_ = _v_80_;
_t_67_ -= _v_77_;
_t_67_ += q_4[k+3][j][i];
_t_67_ -= q_4[k-1][j][i];
_t_65_ -= 0.2 * _t_67_;
double _t_68_ = _v_82_;
_t_68_ -= _v_79_;
_t_68_ += q_4[k+4][j][i];
_t_68_ -= q_4[k-2][j][i];
_t_65_ += 0.038 * _t_68_;
double _t_66_ = _v_78_;
_t_66_ -= _v_86_;
_t_66_ += q_4[k+2][j][i];
_t_66_ -= q_4[k][j][i];
_t_65_ += 0.8 * _t_66_;
flux_3kp1jc0ic0 -= _t_65_ * dxinv2;

flux3_c = flux_3[k+2][j][i];
double flux_3kp2jc0ic0 = flux3_c;
double _t_71_ = q_4[k+3][j][i];
_t_71_ -= q_4[k+1][j][i];
double _t_74_ = -(q_4[k-2][j][i]);
double _t_72_ = q_4[k+4][j][i];
_t_72_ -= q_4[k][j][i];
double _t_73_ = q_4[k+5][j][i];
_t_73_ -= q_4[k-1][j][i];
double _t_76_ = q_4[k+4][j][i];
_t_76_ -= q_4[k+2][j][i];
double _t_77_ = q_4[k+5][j][i];
_t_77_ -= q_4[k+1][j][i];
double _t_78_ = -(q_4[k][j][i]);
double _t_79_ = -(q_4[k-1][j][i]);

_t_71_ += _v_80_;
_t_71_ -= _v_76_;
double _t_70_ = 0.8 * _t_71_;
_t_74_ -= _v_79_;
_t_74_ += _v_100_;
_t_74_ += q_4[k+6][j][i];
_t_78_ += q_4[k+6][j][i];
_t_70_ -= 0.0035 * _t_74_;
_t_72_ += _v_82_;
_t_72_ -= _v_86_;
_t_70_ -= 0.2 * _t_72_;
_t_73_ += _v_91_;
_t_73_ -= _v_77_;
_t_70_ += 0.038 * _t_73_;
flux_3kp2jc0ic0 -= _t_70_ * dxinv2;

flux3_d = flux_3[k+3][j][i];
double flux_3kp3jc0ic0 = flux3_d;

_t_76_ += _v_82_;
_t_76_ -= _v_78_;
double _t_75_ = 0.8 * _t_76_;
_t_77_ += _v_91_;
_t_77_ -= _v_76_;
_t_75_ -= 0.2 * _t_77_;
_t_78_ += _v_100_;
_t_78_ -= _v_86_;
_t_75_ += 0.038 * _t_78_;
_t_79_ += _v_109_;
_t_79_ -= _v_77_;
_t_79_ += q_4[k+7][j][i];
_t_75_ -= 0.0035 * _t_79_;
flux_3kp3jc0ic0 -= _t_75_ * dxinv2;

flux_0[k][j][i] = flux_0kc0jc0ic0;
flux_0[k+1][j][i] = flux_0kp1jc0ic0;
flux_0[k+2][j][i] = flux_0kp2jc0ic0;
flux_0[k+3][j][i] = flux_0kp3jc0ic0;
flux_1[k][j][i] = flux_1kc0jc0ic0;
flux_1[k+1][j][i] = flux_1kp1jc0ic0;
flux_1[k+2][j][i] = flux_1kp2jc0ic0;
flux_1[k+3][j][i] = flux_1kp3jc0ic0;
flux_2[k][j][i] = flux_2kc0jc0ic0;
flux_2[k+1][j][i] = flux_2kp1jc0ic0;
flux_2[k+2][j][i] = flux_2kp2jc0ic0;
flux_2[k+3][j][i] = flux_2kp3jc0ic0;
flux_3[k][j][i] = flux_3kc0jc0ic0;
flux_3[k+1][j][i] = flux_3kp1jc0ic0;
flux_3[k+2][j][i] = flux_3kp2jc0ic0;
flux_3[k+3][j][i] = flux_3kp3jc0ic0;
	} 
}

__global__ void hypterm_2 (double * __restrict__ flux_in_4, double * __restrict__ cons_in_4, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(2*blockdim_k);
	int k = max (k0, 0) + (int)(2*threadIdx.z);

	double (*flux_4)[308][308] = (double (*)[308][308])flux_in_4;
	double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
	double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
	double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
	double (*q_4)[308][308] = (double (*)[308][308])q_in_4;
	double (*cons_4)[308][308] = (double (*)[308][308])cons_in_4;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
		flux_4[k][j][i] = ((0.8*(cons_4[k][j][i+1]*q_1[k][j][i+1]-cons_4[k][j][i-1]*q_1[k][j][i-1]+(q_4[k][j][i+1]*q_1[k][j][i+1]-q_4[k][j][i-1]*q_1[k][j][i-1]))-0.2*(cons_4[k][j][i+2]*q_1[k][j][i+2]-cons_4[k][j][i-2]*q_1[k][j][i-2]+(q_4[k][j][i+2]*q_1[k][j][i+2]-q_4[k][j][i-2]*q_1[k][j][i-2]))+0.038*(cons_4[k][j][i+3]*q_1[k][j][i+3]-cons_4[k][j][i-3]*q_1[k][j][i-3]+(q_4[k][j][i+3]*q_1[k][j][i+3]-q_4[k][j][i-3]*q_1[k][j][i-3]))-0.0035*(cons_4[k][j][i+4]*q_1[k][j][i+4]-cons_4[k][j][i-4]*q_1[k][j][i-4]+(q_4[k][j][i+4]*q_1[k][j][i+4]-q_4[k][j][i-4]*q_1[k][j][i-4])))*dxinv0);
		flux_4[k+1][j][i] = ((0.8*(cons_4[k+1][j][i+1]*q_1[k+1][j][i+1]-cons_4[k+1][j][i-1]*q_1[k+1][j][i-1]+(q_4[k+1][j][i+1]*q_1[k+1][j][i+1]-q_4[k+1][j][i-1]*q_1[k+1][j][i-1]))-0.2*(cons_4[k+1][j][i+2]*q_1[k+1][j][i+2]-cons_4[k+1][j][i-2]*q_1[k+1][j][i-2]+(q_4[k+1][j][i+2]*q_1[k+1][j][i+2]-q_4[k+1][j][i-2]*q_1[k+1][j][i-2]))+0.038*(cons_4[k+1][j][i+3]*q_1[k+1][j][i+3]-cons_4[k+1][j][i-3]*q_1[k+1][j][i-3]+(q_4[k+1][j][i+3]*q_1[k+1][j][i+3]-q_4[k+1][j][i-3]*q_1[k+1][j][i-3]))-0.0035*(cons_4[k+1][j][i+4]*q_1[k+1][j][i+4]-cons_4[k+1][j][i-4]*q_1[k+1][j][i-4]+(q_4[k+1][j][i+4]*q_1[k+1][j][i+4]-q_4[k+1][j][i-4]*q_1[k+1][j][i-4])))*dxinv0);
		flux_4[k][j][i] -= (0.8*(cons_4[k][j+1][i]*q_2[k][j+1][i]-cons_4[k][j-1][i]*q_2[k][j-1][i]+(q_4[k][j+1][i]*q_2[k][j+1][i]-q_4[k][j-1][i]*q_2[k][j-1][i]))-0.2*(cons_4[k][j+2][i]*q_2[k][j+2][i]-cons_4[k][j-2][i]*q_2[k][j-2][i]+(q_4[k][j+2][i]*q_2[k][j+2][i]-q_4[k][j-2][i]*q_2[k][j-2][i]))+0.038*(cons_4[k][j+3][i]*q_2[k][j+3][i]-cons_4[k][j-3][i]*q_2[k][j-3][i]+(q_4[k][j+3][i]*q_2[k][j+3][i]-q_4[k][j-3][i]*q_2[k][j-3][i]))-0.0035*(cons_4[k][j+4][i]*q_2[k][j+4][i]-cons_4[k][j-4][i]*q_2[k][j-4][i]+(q_4[k][j+4][i]*q_2[k][j+4][i]-q_4[k][j-4][i]*q_2[k][j-4][i])))*dxinv1;
		flux_4[k+1][j][i] -= (0.8*(cons_4[k+1][j+1][i]*q_2[k+1][j+1][i]-cons_4[k+1][j-1][i]*q_2[k+1][j-1][i]+(q_4[k+1][j+1][i]*q_2[k+1][j+1][i]-q_4[k+1][j-1][i]*q_2[k+1][j-1][i]))-0.2*(cons_4[k+1][j+2][i]*q_2[k+1][j+2][i]-cons_4[k+1][j-2][i]*q_2[k+1][j-2][i]+(q_4[k+1][j+2][i]*q_2[k+1][j+2][i]-q_4[k+1][j-2][i]*q_2[k+1][j-2][i]))+0.038*(cons_4[k+1][j+3][i]*q_2[k+1][j+3][i]-cons_4[k+1][j-3][i]*q_2[k+1][j-3][i]+(q_4[k+1][j+3][i]*q_2[k+1][j+3][i]-q_4[k+1][j-3][i]*q_2[k+1][j-3][i]))-0.0035*(cons_4[k+1][j+4][i]*q_2[k+1][j+4][i]-cons_4[k+1][j-4][i]*q_2[k+1][j-4][i]+(q_4[k+1][j+4][i]*q_2[k+1][j+4][i]-q_4[k+1][j-4][i]*q_2[k+1][j-4][i])))*dxinv1;
		flux_4[k][j][i] -= (0.8*(cons_4[k+1][j][i]*q_3[k+1][j][i]-cons_4[k-1][j][i]*q_3[k-1][j][i]+(q_4[k+1][j][i]*q_3[k+1][j][i]-q_4[k-1][j][i]*q_3[k-1][j][i]))-0.2*(cons_4[k+2][j][i]*q_3[k+2][j][i]-cons_4[k-2][j][i]*q_3[k-2][j][i]+(q_4[k+2][j][i]*q_3[k+2][j][i]-q_4[k-2][j][i]*q_3[k-2][j][i]))+0.038*(cons_4[k+3][j][i]*q_3[k+3][j][i]-cons_4[k-3][j][i]*q_3[k-3][j][i]+(q_4[k+3][j][i]*q_3[k+3][j][i]-q_4[k-3][j][i]*q_3[k-3][j][i]))-0.0035*(cons_4[k+4][j][i]*q_3[k+4][j][i]-cons_4[k-4][j][i]*q_3[k-4][j][i]+(q_4[k+4][j][i]*q_3[k+4][j][i]-q_4[k-4][j][i]*q_3[k-4][j][i])))*dxinv2;
		flux_4[k+1][j][i] -= (0.8*(cons_4[k+1+1][j][i]*q_3[k+1+1][j][i]-cons_4[k+1-1][j][i]*q_3[k+1-1][j][i]+(q_4[k+1+1][j][i]*q_3[k+1+1][j][i]-q_4[k+1-1][j][i]*q_3[k+1-1][j][i]))-0.2*(cons_4[k+1+2][j][i]*q_3[k+1+2][j][i]-cons_4[k+1-2][j][i]*q_3[k+1-2][j][i]+(q_4[k+1+2][j][i]*q_3[k+1+2][j][i]-q_4[k+1-2][j][i]*q_3[k+1-2][j][i]))+0.038*(cons_4[k+1+3][j][i]*q_3[k+1+3][j][i]-cons_4[k+1-3][j][i]*q_3[k+1-3][j][i]+(q_4[k+1+3][j][i]*q_3[k+1+3][j][i]-q_4[k+1-3][j][i]*q_3[k+1-3][j][i]))-0.0035*(cons_4[k+1+4][j][i]*q_3[k+1+4][j][i]-cons_4[k+1-4][j][i]*q_3[k+1-4][j][i]+(q_4[k+1+4][j][i]*q_3[k+1+4][j][i]-q_4[k+1-4][j][i]*q_3[k+1-4][j][i])))*dxinv2;
	} 
}

extern "C" void host_code (double *h_flux_0, double *h_flux_1, double *h_flux_2, double *h_flux_3, double *h_flux_4, double *h_cons_1, double *h_cons_2, double *h_cons_3, double *h_cons_4, double *h_q_1, double *h_q_2, double *h_q_3, double *h_q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	double *flux_0;
	hipMalloc (&flux_0, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_0\n");
	hipMemcpy (flux_0, h_flux_0, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_1;
	hipMalloc (&flux_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_1\n");
	hipMemcpy (flux_1, h_flux_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_2;
	hipMalloc (&flux_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_2\n");
	hipMemcpy (flux_2, h_flux_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_3;
	hipMalloc (&flux_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_3\n");
	hipMemcpy (flux_3, h_flux_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_4;
	hipMalloc (&flux_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_4\n");
	hipMemcpy (flux_4, h_flux_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_1;
	hipMalloc (&cons_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_1\n");
	hipMemcpy (cons_1, h_cons_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_2;
	hipMalloc (&cons_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_2\n");
	hipMemcpy (cons_2, h_cons_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_3;
	hipMalloc (&cons_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_3\n");
	hipMemcpy (cons_3, h_cons_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_4;
	hipMalloc (&cons_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_4\n");
	hipMemcpy (cons_4, h_cons_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_1;
	hipMalloc (&q_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_1\n");
	hipMemcpy (q_1, h_q_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_2;
	hipMalloc (&q_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_2\n");
	hipMemcpy (q_2, h_q_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_3;
	hipMalloc (&q_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_3\n");
	hipMemcpy (q_3, h_q_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_4;
	hipMalloc (&q_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_4\n");
	hipMemcpy (q_4, h_q_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 4, 4);
	dim3 gridconfig_0 (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, blockconfig.z));
	hypterm_0 <<<gridconfig_0, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, cons_1, cons_2, cons_3, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);
	dim3 gridconfig_1 (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, 4*blockconfig.z));
	hypterm_1 <<<gridconfig_1, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, cons_1, cons_2, cons_3, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);
	dim3 gridconfig_2 (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, 2*blockconfig.z));
	hypterm_2 <<<gridconfig_2, blockconfig>>> (flux_4, cons_4, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);

	hipMemcpy (h_flux_0, flux_0, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_1, flux_1, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_3, flux_3, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_4, flux_4, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_2, flux_2, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
}
